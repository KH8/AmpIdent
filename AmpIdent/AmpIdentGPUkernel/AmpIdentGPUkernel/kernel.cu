#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <>
#include <math.h>
 
__global__ void kernel(float* a, float* b, float* out, int cola, int colb, int rowa)
{
	long i = blockDim.x * blockIdx.x + threadIdx.x;
	
	long z = (int)floor((float)(i / (colb)));
	long y = i - z*colb;

	for (int x = 0; x < cola; x++)
	{
		if (i < rowa * colb)
		{
			out[z*colb + y] += a[z*cola + x] * b[x*colb + y];
		}
	}
}
 
// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 1; i <= 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i-1]);
    for (int i = 1; i <= 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i-1]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %u\n",  devProp.totalConstMem);
    printf("Texture alignment:             %u\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}
 
int main()
{
	/*
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);
 
    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }
 
    printf("\nPress any key to exit...");
    char c;
    scanf("%c", &c);
	//*/

    return 0;
}